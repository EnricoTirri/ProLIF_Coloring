#include "hip/hip_runtime.h"

#include "Mesh.hpp"

__global__
void addMask_ker(MoleculeMesh::data_t *data, const MoleculeMesh::data_t *to_add,
                 const int displ_x, const int displ_y, const int displ_z,
                 const int data_dim_x, const int data_dim_y, const int data_dim_z,
                 const int add_dim_x, const int add_dim_y, const int add_dim_z) {

    const int thr_id = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x);


    const int data_layer_size = data_dim_x * data_dim_y;
    const int data_z = thr_id / data_layer_size;

    if (data_z < data_dim_z) {
        const int data_y = (thr_id % data_layer_size) / data_dim_y;
        const int data_x = (thr_id % data_layer_size) % data_dim_y;

        const int add_z = data_z - displ_z;
        const int add_y = data_y - displ_y;
        const int add_x = data_x - displ_x;

        if (add_z >= 0 && add_y >= 0 && add_x >= 0 && add_z < add_dim_z && add_y < add_dim_y && add_x < add_dim_x) {
            const int data_id = data_dim_x * (data_z * data_dim_y + data_y) + data_x;
            const int add_id = add_dim_x * (add_z * add_dim_y + add_y) + add_x;
            data[data_id] = data[data_id] || to_add[add_id];
        }
    }
}

void MoleculeMesh::addMeshes(MoleculeMesh::data_t *data, MoleculeMesh::data_t *to_add,
                             const int displ_x, const int displ_y, const int displ_z,
                             const int data_dim_x, const int data_dim_y, const int data_dim_z,
                             const int add_dim_x, const int add_dim_y, const int add_dim_z) {

    int dataDim = data_dim_x * data_dim_y * data_dim_z;
    unsigned int numBlocks = (dataDim + BLOCK_SIZE) / (BLOCK_SIZE);

    addMask_ker<<<numBlocks, BLOCK_SIZE>>>(data, to_add, displ_x, displ_y, displ_z,
                                           data_dim_x, data_dim_y, data_dim_z,
                                           add_dim_x, add_dim_y, add_dim_z);
}

__global__
void subMask_ker(MoleculeMesh::data_t *data, const MoleculeMesh::data_t *to_subtract,
                 const int displ_x, const int displ_y, const int displ_z,
                 const int data_dim_x, const int data_dim_y, const int data_dim_z,
                 const int sub_dim_x, const int sub_dim_y, const int sub_dim_z) {

    int thr_id = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x);

    const int data_layer_size = data_dim_x * data_dim_y;
    const int data_z = thr_id / data_layer_size;

    if (data_z < data_dim_z) {
        const int data_y = (thr_id % data_layer_size) / data_dim_y;
        const int data_x = (thr_id % data_layer_size) % data_dim_y;

        const int sub_z = data_z + displ_z;
        const int sub_y = data_y + displ_y;
        const int sub_x = data_x + displ_x;

        if (sub_z >= 0 && sub_y >= 0 && sub_x >= 0 && sub_z < sub_dim_z && sub_y < sub_dim_y && sub_x < sub_dim_x) {
            data[thr_id] = data[thr_id] && !to_subtract[sub_dim_x * (sub_z * sub_dim_y + sub_y) + sub_x];
        }
    }
}

void MoleculeMesh::subMeshes(MoleculeMesh::data_t *data, MoleculeMesh::data_t *to_subtract,
                             const int displ_x, const int displ_y, const int displ_z,
                             const int data_dim_x, const int data_dim_y, const int data_dim_z,
                             const int sub_dim_x, const int sub_dim_y, const int sub_dim_z) {

    int dataDim = data_dim_x * data_dim_y * data_dim_z;
    unsigned int numBlocks = (dataDim + BLOCK_SIZE) / (BLOCK_SIZE);

    subMask_ker<<<numBlocks, BLOCK_SIZE>>>(data, to_subtract, displ_x, displ_y, displ_z,
                                           data_dim_x, data_dim_y, data_dim_z,
                                           sub_dim_x, sub_dim_y, sub_dim_z);
}
