#include "hip/hip_runtime.h"

#include <GraphMol/FileParsers/FileParsers.h>
#include "DistanceInteraction.hpp"
#include "Discretizer.hpp"

__global__
void buildBubble_ker(MoleculeMesh::data_t *bubble, const double inter_d, const int maskEdge) {
    int thr_id = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x);
    const int layerDim = maskEdge * maskEdge;

    if (thr_id < layerDim * maskEdge) {
        const int z_cord = thr_id / layerDim;
        const int y_cord = (thr_id % layerDim) / maskEdge;
        const int x_cord = (thr_id % layerDim) % maskEdge;

        const int maskRadius = maskEdge / 2;

        // Over all size of pattern-mesh assign if (point-distance <= #distance) from the center of mesh
        double ds = inter_d * inter_d;
        int dz = z_cord - maskRadius;
        int z_res = dz * dz;
        int dy = y_cord - maskRadius;
        int y_res = dy * dy;
        int dx = x_cord - maskRadius;
        int x_res = dx * dx;

        bubble[thr_id] = (x_res + y_res + z_res <= ds);
    }
}


bool DistanceInteraction::getInteraction(const RDKit::ROMol *molecule, MoleculeMesh &interactionMask,
                                         MoleculeMesh &subtractionMask) {
    hipError_t err;
    MoleculeMesh::data_t *bubble_data = nullptr;
    MoleculeMesh::data_t *interaction_data = nullptr;
    MoleculeMesh::data_t *subtraction_data = nullptr;
    bool ris = false;

    try {
        // Get molecule conformer and retrive matches of smart into given molecule
        RDKit::Conformer conformer = molecule->getConformer();
        std::vector<RDKit::MatchVectType> *matches = Interaction::findMatch(molecule);

        if (matches->empty()) return false;

        // Discretize mask radius and calculate mask dimension
        // Over all size of pattern-mesh assign if (point-distance <= #distance) from the center of mesh
        double scaledDistance = distance * GRAIN;
        int scaledMaskRadius = static_cast<int>(ceil(scaledDistance));
        int maskDim = 2 * scaledMaskRadius;

        int bubbleDim = maskDim * maskDim * maskDim;

        err = hipMalloc((void **) &bubble_data, sizeof(MoleculeMesh::data_t) * (bubbleDim));
        if (err != hipSuccess) throw;

        unsigned int numBlocks = (interactionMask.getDataSize() + BLOCK_SIZE) / BLOCK_SIZE;


        buildBubble_ker<<<numBlocks, BLOCK_SIZE>>>(bubble_data, scaledDistance, maskDim);
        err = hipGetLastError();
        if (err != hipSuccess) throw;


        err = hipMalloc((void **) &interaction_data, sizeof(MoleculeMesh::data_t) * interactionMask.getDataSize());
        if (err != hipSuccess) throw;

        // For each interaction-centroid apply the pattern-mesh centered at centroid onto the support-mesh
        auto paddingDisplacement = static_cast<double>(interactionMask.internalDisplacement - scaledMaskRadius);

        for (RDKit::MatchVectType match: *matches) {
            if (!match.empty()) {
                ris = true;
                // Get interaction match centroid position
                auto atomId = match.at(0).second;
                RDGeom::Point3D pos = conformer.getAtomPos(atomId);

                // Find the zero-point displacement of pattern from the zero-point of support-mask
                double px = (pos.x - interactionMask.globalDisplacement.x) * GRAIN + paddingDisplacement;
                double py = (pos.y - interactionMask.globalDisplacement.y) * GRAIN + paddingDisplacement;
                double pz = (pos.z - interactionMask.globalDisplacement.z) * GRAIN + paddingDisplacement;

                // Discretize the displacement
                int displ_x = static_cast<int>(round(px));
                int displ_y = static_cast<int>(round(py));
                int displ_z = static_cast<int>(round(pz));

                MoleculeMesh::addMeshes(interaction_data, bubble_data,
                                        displ_x, displ_y, displ_z,
                                        interactionMask.dim_x, interactionMask.dim_y, interactionMask.dim_z,
                                        maskDim, maskDim, maskDim);
                err = hipGetLastError();
                if (err != hipSuccess) throw;
            }
        }

        if (subtractionMask.getDataSize()!=0) {
            err = hipMalloc((void **) &subtraction_data, sizeof(MoleculeMesh::data_t) * subtractionMask.getDataSize());
            if (err != hipSuccess) throw;

            err = hipMemcpy(subtraction_data, subtractionMask.getData(),
                             sizeof(MoleculeMesh::data_t) * subtractionMask.getDataSize(), hipMemcpyHostToDevice);
            if (err != hipSuccess) throw;

            MoleculeMesh::subMeshes(interaction_data, subtraction_data,
                                    0, 0, 0,
                                    interactionMask.dim_x, interactionMask.dim_y, interactionMask.dim_z,
                                    subtractionMask.dim_x, subtractionMask.dim_y, subtractionMask.dim_z);
            err = hipGetLastError();
            if (err != hipSuccess) throw;
        }


        err = hipMemcpy(interactionMask.getData(), interaction_data,
                         sizeof(MoleculeMesh::data_t) * interactionMask.getDataSize(), hipMemcpyDeviceToHost);
        if(err != hipSuccess) throw;

    } catch (...) {
        ris = false;
        std::cout << hipGetErrorString(err) << std::endl;
    }

    if(bubble_data != nullptr) hipFree(bubble_data);
    if(interaction_data != nullptr) hipFree(interaction_data);
    if(subtraction_data != nullptr) hipFree(subtraction_data);

    return ris;
}