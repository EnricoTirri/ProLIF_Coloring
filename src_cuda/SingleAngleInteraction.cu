#include "hip/hip_runtime.h"

#include "SingleAngleInteraction.hpp"
#include <cuda/std/cmath>

__device__
double lengthSq(double x, double y, double z) {
    return x * x + y * y + z * z;
}

__device__
double dotProduct(double p1_x, double p1_y, double p1_z,
                  double p2_x, double p2_y, double p2_z) {
    return p1_x * (p2_x) + p1_y * (p2_y) + p1_z * (p2_z);
}

__device__
double angleTo(double p1_x, double p1_y, double p1_z,
               double p2_x, double p2_y, double p2_z) {
    double lsq = lengthSq(p1_x, p1_y, p1_z) * lengthSq(p2_x, p2_y, p2_z);
    double dotProd = dotProduct(p1_x, p1_y, p1_z, p2_x, p2_y, p2_z);
    dotProd /= cuda::std::sqrt(lsq);

    // watch for roundoff error:
    if (dotProd <= -1.0) {
        return M_PI;
    }
    if (dotProd >= 1.0) {
        return 0.0;
    }

    return cuda::std::acos(dotProd);
}

__global__
void buildBubbleSlice_ker(MoleculeMesh::data_t *bubble,
                          const double inter_d, const double min_angle, const double max_angle,
                          const double center_x, const double center_y, const double center_z,
                          const double p1_x, const double p1_y, const double p1_z,
                          const double p2_x, const double p2_y, const double p2_z,
                          const int maskEdge) {

    int thr_id = static_cast<int>(blockIdx.x * blockDim.x + threadIdx.x);
    const int layerDim = maskEdge * maskEdge;

    if (thr_id < layerDim * maskEdge) {

        const int z_cord = thr_id / layerDim;
        const int y_cord = (thr_id % layerDim) / maskEdge;
        const int x_cord = (thr_id % layerDim) % maskEdge;

        const int maskRadius = maskEdge / 2;

        // Over all size of pattern-mesh assign if (point-distance <= #distance) from the center of mesh

        // Calculate vector p2 --> p1
        double p2p1_x = p1_x - p2_x;
        double p2p1_y = p1_y - p2_y;
        double p2p1_z = p1_z - p2_z;

        double p2p1_l = cuda::std::sqrt(lengthSq(p2p1_x, p2p1_y, p2p1_z));
        p2p1_x /= p2p1_l;
        p2p1_y /= p2p1_l;
        p2p1_z /= p2p1_l;

        /*
         * Over all size of pattern-mesh assign if:
         *      - (point-distance <= #distance) from the center of mesh
         *      - angle between l1 <-- p2 --> p1 is (#min <= #angle <= #max)
         */
        double ds = inter_d * inter_d;
        int dz = z_cord - maskRadius;
        int z_res = dz * dz;
        int dy = y_cord - maskRadius;
        int y_res = dy * dy;
        int dx = x_cord - maskRadius;
        int x_res = dx * dx;

        // Position of l1 is calculated taking account of pattern-center position

        double l1_z = z_cord + center_z;
        double l1_y = y_cord + center_y;
        double l1_x = x_cord + center_x;

        // Calculate vector p2 --> l1
        double p2l1_x = l1_x - p2_x;
        double p2l1_y = l1_y - p2_y;
        double p2l1_z = l1_z - p2_z;

        double p2l1_l = cuda::std::sqrt(lengthSq(p2l1_x, p2l1_y, p2l1_z));
        p2l1_x /= p2l1_l;
        p2l1_y /= p2l1_l;
        p2l1_z /= p2l1_l;

        // Calculate angle l1 <-- p2 --> p1
        double angle = angleTo(p2p1_x, p2p1_y, p2p1_z, p2l1_x, p2l1_y, p2l1_z);

        bubble[thr_id] = (x_res + y_res + z_res <= ds && angle >= min_angle && angle <= max_angle);
    }
}

bool SingleAngleInteraction::getInteraction(const RDKit::ROMol *molecule,
                                            MoleculeMesh &interactionMask, MoleculeMesh &subtractionMask) {
    hipError_t err;
    MoleculeMesh::data_t *bubble_data = nullptr;
    MoleculeMesh::data_t *interaction_data = nullptr;
    MoleculeMesh::data_t *subtraction_data = nullptr;
    bool ris = false;

    try {
        // Get molecule conformer and retrive matches of smart into given molecule
        RDKit::Conformer conformer = molecule->getConformer();
        std::vector<RDKit::MatchVectType> *matches = Interaction::findMatch(molecule);

        if (matches->empty()) return false;

        // Calculate mask size and centering coordinates
        double scaledDistance = distance * GRAIN;
        auto scaledMaskCenter = static_cast<int>(ceil(scaledDistance));
        int maskDim = 2 * scaledMaskCenter;


        int bubbleDim = maskDim * maskDim * maskDim;

        unsigned int numBlocks = (interactionMask.getDataSize() + BLOCK_SIZE) / BLOCK_SIZE;

        err = hipMalloc((void **) &bubble_data, sizeof(MoleculeMesh::data_t) * (bubbleDim));
        if (err != hipSuccess) throw;

        err = hipMalloc((void **) &interaction_data, sizeof(MoleculeMesh::data_t) * interactionMask.getDataSize());
        if (err != hipSuccess) throw;

        for (RDKit::MatchVectType match: *matches) {
            if (match.size() >= 2) {
                ris = true;

                // Get molecule match and its centroids position
                auto p1Id = match.at(0).second;
                auto p1 = conformer.getAtomPos(p1Id);
                auto p2Id = match.at(1).second;
                auto p2 = conformer.getAtomPos(p2Id);

                RDGeom::Point3D center;
                if (cp) center = p1;
                else center = p2;

                buildBubbleSlice_ker<<<numBlocks, BLOCK_SIZE>>>(bubble_data,
                                                                scaledDistance, min_angle, max_angle,
                                                                center.x, center.y, center.z,
                                                                p1.x, p1.y, p1.z,
                                                                p2.x, p2.y, p2.z,
                                                                maskDim);
                err = hipGetLastError();
                if (err != hipSuccess) throw;

                // Find the zero-point displacement of pattern from the zero-point of support-mask
                auto paddingDisplacement = static_cast<double>(interactionMask.internalDisplacement - scaledMaskCenter);
                double px = (center.x - interactionMask.globalDisplacement.x) * GRAIN + paddingDisplacement;
                double py = (center.y - interactionMask.globalDisplacement.y) * GRAIN + paddingDisplacement;
                double pz = (center.z - interactionMask.globalDisplacement.z) * GRAIN + paddingDisplacement;

                // Discretize the displacement
                int displ_x = static_cast<int>(round(px));
                int displ_y = static_cast<int>(round(py));
                int displ_z = static_cast<int>(round(pz));

                // Apply pattern at displacement onto support-mesh
                MoleculeMesh::addMeshes(interaction_data, bubble_data,
                                        displ_x, displ_y, displ_z,
                                        interactionMask.dim_x, interactionMask.dim_y, interactionMask.dim_z,
                                        maskDim, maskDim, maskDim);
                err = hipGetLastError();
                if (err != hipSuccess) throw;
            }
        }

        if (!subtractionMask.getDataSize()!=0) {
            err = hipMalloc((void **) &subtraction_data, sizeof(MoleculeMesh::data_t) * subtractionMask.getDataSize());
            if (err != hipSuccess) throw;

            err = hipMemcpy(subtraction_data, subtractionMask.getData(),
                             sizeof(MoleculeMesh::data_t) * subtractionMask.getDataSize(), hipMemcpyHostToDevice);
            if (err != hipSuccess) throw;

            MoleculeMesh::subMeshes(interaction_data, subtraction_data,
                                    0, 0, 0,
                                    interactionMask.dim_x, interactionMask.dim_y, interactionMask.dim_z,
                                    subtractionMask.dim_x, subtractionMask.dim_y, subtractionMask.dim_z);
            err = hipGetLastError();
            if (err != hipSuccess) throw;
        }

        err = hipMemcpy(interactionMask.getData(), interaction_data,
                         sizeof(MoleculeMesh::data_t) * interactionMask.getDataSize(), hipMemcpyDeviceToHost);
        if (err != hipSuccess) throw;

    } catch (...) {
        ris = false;
        std::cout << hipGetErrorString(err) << std::endl;
    }

    if (bubble_data != nullptr) hipFree(bubble_data);
    if (interaction_data != nullptr) hipFree(interaction_data);
    if (subtraction_data != nullptr) hipFree(subtraction_data);

    return ris;
}